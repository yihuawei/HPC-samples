

#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 16

#define WA (32 * BLOCK_SIZE) // Matrix A width
#define HA (16 * BLOCK_SIZE) // Matrix A height
#define WB (24 * BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height


#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]

__global__ void
matrixMul( float* C, float* A, float* B, int wA, int wB)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int aBegin = wA * BLOCK_SIZE * by;

    int aEnd   = aBegin + wA - 1;

    int aStep  = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;

    int bStep  = BLOCK_SIZE * wB;

    float Csub = 0;

    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {


        AS(ty, tx) = A[a + wA * ty + tx];
        BS(tx, ty) = B[b + wB * tx + ty];

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += AS(ty, k) * BS(k, tx);

        __syncthreads();
    }

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

int main(){
 
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);
    
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);    

    dim3 threads,grid;
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
    return 0;


}
